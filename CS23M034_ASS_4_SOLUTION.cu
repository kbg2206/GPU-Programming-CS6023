#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//******************************************

// Write down the kernels here

__global__ void kernel3(int *Health,int *ctr)
{
  //printf("%d\n",threadIdx.x);
  if(Health[threadIdx.x]>0)
  {
    atomicAdd(ctr,1);
  }

  
}

__global__ void kernel2(int *copy,int *score,int *Xcoor,int *Ycoor,int *Health,int T,long int k)
{
  //*ctr = 0;

  if(k%T == 0 || copy[blockIdx.x]<=0) return ;


  __shared__ long long MinDis;
  MinDis = LLONG_MAX;

  __syncthreads();

  int src= blockIdx.x;
  int des = threadIdx.x;
  int direction = (src+k)%T;

  int dirXcoor = Xcoor[direction] - Xcoor[src];
  int dirYcoor = Ycoor[direction] - Ycoor[src];
  int desXcoor = Xcoor[des] - Xcoor[src];
  int desYcoor = Ycoor[des] - Ycoor[src];

  //printf("%d %d %d %d %d %d %d\n",src,des,direction,dirXcoor,dirYcoor,desXcoor,desYcoor);

  int lhs = desXcoor*dirYcoor;
  int rhs = desYcoor*dirXcoor;
  long long distance =(long long) desYcoor * desYcoor + (long long) desXcoor * desXcoor;
  long int QuadX  = (long int)desXcoor * dirXcoor;
  long int QuadY  = (long int)desYcoor * dirYcoor;

  // if (dirXcoor>=0 && dirYcoor>=0) dirQuad=1;
  // else if (dirXcoor<0 && dirYcoor>=0) dirQuad=2;
  // else if (dirXcoor<0 && dirYcoor<0) dirQuad=3;
  // else dirQuad = 4;

  // if (desXcoor>=0 && desYcoor>=0) desQuad=1;
  // else if (desXcoor<0 && desYcoor>=0) desQuad=2;
  // else if (desXcoor<0 && desYcoor<0) desQuad=3;
  // else desQuad = 4;

  

  //printf("%d %d   ->   %d %d %ld %d %d\n",src,des,lhs,rhs,distance,dirQuad,desQuad);

  

  if(distance!=0  && copy[des]>0 && lhs == rhs && QuadX >=0 && QuadY>=0)
  {
    atomicMin(&MinDis,distance);
  }

  __syncthreads();

  //printf("%d %d  ->  %ld %ld %d %d\n",src,des,MinDis,distance,dirQuad,desQuad);

  //printf("Hello\n");
  if(MinDis == distance && QuadX >=0 && QuadY>=0 && lhs == rhs  && copy[des]>0)
  {
    
    atomicAdd(Health+des,-1);
    score[src]++;
    //atomicAdd(score+src,1);
  }

 // printf("%d %d  ->  %d %d %d\n",src,des,Health[des],score[src],MinDis);

  
}

__global__ void kernel1(int *Score,int *Health,int H)
{
   Health[threadIdx.x] = H;
   Score[threadIdx.x] = 0;
}



//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************



    


    int *GpuScore;
    hipMalloc(&GpuScore,sizeof(int)*T);
    

    int *GpuHealth;
    hipMalloc(&GpuHealth,sizeof(int)*T);

    // -------- Kernel Launch for Initialization -------------
    kernel1<<<1,T>>>(GpuScore,GpuHealth,H);

    int *GpuHealthcopy;
    hipMalloc(&GpuHealthcopy,sizeof(int)*T);

    int *GpuXcoor;
    hipMalloc(&GpuXcoor,sizeof(int)*T);
    hipMemcpy(GpuXcoor,xcoord,sizeof(int)*T,hipMemcpyHostToDevice);

    int *GpuYcoor;
    hipMalloc(&GpuYcoor,sizeof(int)*T);
    hipMemcpy(GpuYcoor,ycoord,sizeof(int)*T,hipMemcpyHostToDevice);

    int *ctr;
    hipHostAlloc(&ctr,sizeof(int),0);
    
    
    long int k=1;
    //printf("%ld\n",k);
  
    do
    { 
       *ctr = 0;
       
       hipMemcpy(GpuHealthcopy,GpuHealth,sizeof(int)*T,hipMemcpyDeviceToDevice);
       kernel2<<<T,T>>>(GpuHealthcopy,GpuScore,GpuXcoor,GpuYcoor,GpuHealth,T,k);
       kernel3<<<1,T>>>(GpuHealth,ctr);
       hipDeviceSynchronize();
       //cout<<k<<" "<<*ctr<<"\n";
       //if(k==1000) break;
       
       k++;
    }while(*ctr>=2 );
    

    
  
    



    hipMemcpy(score,GpuScore,sizeof(int)*T,hipMemcpyDeviceToHost);

    hipFree(GpuScore);
    hipFree(GpuHealth);
    hipFree(GpuHealthcopy);
    hipFree(GpuXcoor);
    hipFree(GpuYcoor);
    hipHostFree(ctr);
    // cudaFreeHost(k);

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}