
/**
*   CS6023: GPU Programming
*   Assignment 2
*
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree()
*   to free up memory as soon as you're done with an allocation.
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void dkernel(long int *matrix,long int *filter,long int *result,int m,int n,int k,int chunkSize)
{
  extern __shared__ long int arr[];
  long int id = blockIdx.x * blockDim.x + threadIdx.x;


  for(int i=0;i<chunkSize;++i)
  {
    if(threadIdx.x+i*n < k*k)
    {
       arr[threadIdx.x+i*n]=filter[threadIdx.x+i*n];
    }
  }

  __syncthreads();



  long int row=id/n;
  long int col=id%n;


  for(int i=row-k/2;i<=row+k/2;++i)
  {
    for(int j=col-k/2;j<=col+k/2;++j)
    {
      int filter_row=i-row+k/2;
      int filter_col=j-col+k/2;
      //printf("%d %d %d %d\n",i,j,filter_row,filter_col);
      if(i>=0 && i<m && j>=0 && j<n)
      {
        result[id]+=filter[filter_row*k+filter_col]*matrix[i*n+j];
      }
    }
  }


}



int main(int argc, char** argv) {

  //freopen("test2.txt","r",stdin);
  //freopen("output.txt","w",stdout);
    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];


    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
    **/

    long int *gmatrix;
    long int *gfilter;
    long int *result;
    int chunkSize;
    chunkSize=ceil((1.0*k*k)/n);
    hipMalloc(&gmatrix,sizeof(long int)*m*n);
    hipMemcpy(gmatrix,h_mat,sizeof(long int)*m*n,hipMemcpyHostToDevice);



    hipMalloc(&result,sizeof(long int)*m*n);
    hipMemset(result,0,sizeof(long int)*m*n);


    hipMalloc(&gfilter,sizeof(long int)*k*k);
    hipMemcpy(gfilter,h_filter,sizeof(long int)*k*k,hipMemcpyHostToDevice);
    /****************************************************Start Here***********************************************************/

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch

    dkernel<<<m,n,sizeof(long int)*k*k>>>(gmatrix,gfilter,result,m,n,k,chunkSize);
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    hipMemcpy(h_ans,result,sizeof(long int)*m*n,hipMemcpyDeviceToHost);


     hipFree(gmatrix);
     hipFree(gfilter);
     hipFree(result);

     
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
    */



    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}
