#include "hip/hip_runtime.h"

/*
	CS 6023 Assignment 3.
	Do not make any changes to the boiler plate code or the other files in the folder.
	Use hipFree to deallocate any memory not in usage.
	Optimize as much as possible.
*/

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>


int ctr=0;
int dfs(int node,int *Offset,int *Csr,int *preorder,int *size_chart,int *start_chart)
{

	int start=Offset[node];
	int end=Offset[node+1];
  preorder[ctr++]=node;
  start_chart[node]=ctr-1;

  size_chart[node]=0;
	for(int i=start;i<end;++i)
	{
		int curr_node=Csr[i];
		size_chart[node]+=dfs(curr_node,Offset,Csr,preorder,size_chart,start_chart);

	}
  size_chart[node]+=(end-start);

  return size_chart[node];
}



__global__ void transaltion_kernel(int *preorder,int *start_chart,int *size_chart,int *transalation,int *CoordinateX,int *CoordinateY ,int n)
{
  long int id = blockIdx.x * blockDim.x + threadIdx.x;


	if(id<n)
	{

			int Mesh_id=transalation[id];
			int direction=transalation[id+n];
			int step=transalation[id+n*2];

			int size=size_chart[Mesh_id];
      int start=start_chart[Mesh_id];

      for(int i=start;i<=start+size;++i)
      {
        int *resultX=CoordinateX+preorder[i];
        int *resultY=CoordinateY+preorder[i];
        int dir[] = {-1,+1};

        if(direction/2==0)
        {
          atomicAdd(resultX,dir[direction]*step);
        }
        else
        {
          atomicAdd(resultY,dir[direction%2]*step);
        }
        
      }

	}
}


__global__ void map_create_kernel(int *map,int *opacity,int size)
{
	long int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<size)
	{
		map[opacity[id]]=id;
	}
}

__global__ void opacity_kernel(int *Opacity,int *MeshSizeX,int *MeshSizeY,int *MeshStartX,int *MeshStartY,int *result,int V,int frameSizeX,int frameSizeY)
{
	long int id = ((long int)blockIdx.x * blockDim.x )+ threadIdx.x;
  int node = id / 10000;
  int idx  = id % 10000;


  if(node < V && idx < MeshSizeX[node]*MeshSizeY[node])
  {
      
      int row = idx / MeshSizeY[node];
      int col = idx % MeshSizeY[node];
      long int scene_idx = MeshStartX[node]+row;
      long int scene_idy = MeshStartY[node]+col;
      if(scene_idx>=0 && scene_idx<frameSizeX && scene_idy >=0 && scene_idy<frameSizeY)
      {
        int scene_id  = scene_idx * frameSizeY + scene_idy;
        atomicMax(&result[scene_id],Opacity[node]);
      }
  }


}




__global__ void scene_kernel(int *map,int *result,int **Mesh,int *MeshSizeX,int *MeshSizeY,int *MeshStartX,int *MeshStartY,int frameSizeX,int frameSizeY)
{
  	long int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id<frameSizeX*frameSizeY)
    {
      int row = id / frameSizeY;
      int col = id % frameSizeY;

      
      if(result[id]!=-1)
      {
        int node = map[result[id]];
        int Mesh_idx = row-MeshStartX[node];
        int Mesh_idy = col-MeshStartY[node];
        int Mesh_id = Mesh_idx*MeshSizeY[node]+Mesh_idy;
        result[id]=Mesh[node][Mesh_id];
      }
      else
      {
        result[id]=0;
      }
      
    }
}
void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input.
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;


	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ;
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ;
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}


void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL;
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}

	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}


int main (int argc, char **argv) {

	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ;
	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;

	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.



auto start = std::chrono::high_resolution_clock::now () ;

  int *preorder;
  int *start_chart;
  int *size_chart;
  preorder=(int * )malloc(sizeof(int)*V);
  start_chart=(int * )malloc(sizeof(int)*V);
  size_chart=(int * )malloc(sizeof(int)*V);
  // ------- dfs call to store graph information -------------------------
  dfs(0,hOffset,hCsr,preorder,size_chart,start_chart);


// ---------------------------- GPU Allocation Start Here -------------------

	int *GpuPreOrder;
  hipMalloc(&GpuPreOrder,sizeof(int)*V);
  hipMemcpy(GpuPreOrder,preorder,sizeof(int)*V,hipMemcpyHostToDevice);


  int *GpuStartChart;
  hipMalloc(&GpuStartChart,sizeof(int)*V);
  hipMemcpy(GpuStartChart,start_chart,sizeof(int)*V,hipMemcpyHostToDevice);


  int *GpuSizeChart;
  hipMalloc(&GpuSizeChart,sizeof(int)*V);
  hipMemcpy(GpuSizeChart,size_chart,sizeof(int)*V,hipMemcpyHostToDevice);


	int *GpuGlobalCoordinatesX;
	hipMalloc(&GpuGlobalCoordinatesX,sizeof(int)*V);
	hipMemcpy(GpuGlobalCoordinatesX,hGlobalCoordinatesX,sizeof(int)*V,hipMemcpyHostToDevice);


	int *GpuGlobalCoordinatesY;
	hipMalloc(&GpuGlobalCoordinatesY,sizeof(int)*V);
	hipMemcpy(GpuGlobalCoordinatesY,hGlobalCoordinatesY,sizeof(int)*V,hipMemcpyHostToDevice);


	int *GpuTranslations;
	int *dummyTranslations=(int *)malloc(sizeof(int)*numTranslations*3);
	for(int i=0;i<numTranslations;++i)
	{
		*(dummyTranslations+i)=translations[i][0];
		*(dummyTranslations+numTranslations+i)=translations[i][1];
		*(dummyTranslations+numTranslations*2+i)=translations[i][2];
    //cout<<*(dummyTranslations+i*numTranslations+0)<<" "<<*(dummyTranslations+i*numTranslations+1)<<" "<<*(dummyTranslations+i*numTranslations+2)<<"\n";
	}
	hipMalloc(&GpuTranslations,sizeof(int)*numTranslations*3);
  hipMemcpy(GpuTranslations,dummyTranslations,sizeof(int)*numTranslations*3,hipMemcpyHostToDevice);



	// -------transalation Kernel launch ---------------
  long int block=ceil((1.0 * numTranslations)/1024);
  transaltion_kernel<<<block,1024>>>(GpuPreOrder,GpuStartChart,GpuSizeChart,GpuTranslations,GpuGlobalCoordinatesX,GpuGlobalCoordinatesY,numTranslations);
  // -------------------------------------------------------

	hipFree(GpuPreOrder);
	hipFree(GpuSizeChart);
	hipFree(GpuStartChart);
	hipFree(GpuTranslations);


	int *GpuOpacity;
	hipMalloc(&GpuOpacity,sizeof(int)*V);
	hipMemcpy(GpuOpacity,hOpacity,sizeof(int)*V,hipMemcpyHostToDevice);


	int *GpuFrameSizeX;
	hipMalloc(&GpuFrameSizeX,sizeof(int)*V);
	hipMemcpy(GpuFrameSizeX,hFrameSizeX,sizeof(int)*V,hipMemcpyHostToDevice);


  int *GpuFrameSizeY;
	hipMalloc(&GpuFrameSizeY,sizeof(int)*V);
	hipMemcpy(GpuFrameSizeY,hFrameSizeY,sizeof(int)*V,hipMemcpyHostToDevice);


	int **GpuMesh;
	int **dummyMesh=(int **)malloc(sizeof(int *)*V);
	for(int i=0;i<V;++i)
	{
		int *address=*(hMesh+i);
		int m=*(hFrameSizeX+i);
		int n=*(hFrameSizeY+i);
		int *Mesh;

		hipMalloc(&Mesh,sizeof(int)*m*n);
		hipMemcpy(Mesh,address,sizeof(int)*m*n,hipMemcpyHostToDevice);
		*(dummyMesh+i)=Mesh;
	}

	hipMalloc(&GpuMesh,sizeof(int *)*V);
	hipMemcpy(GpuMesh,dummyMesh,sizeof(int*)*V,hipMemcpyHostToDevice);


  int *Opacity_matrix;
  hipMalloc(&Opacity_matrix,sizeof(int)*frameSizeX*frameSizeY);
  hipMemset(Opacity_matrix,-1,sizeof(int)*frameSizeX*frameSizeY);



  // ---------------------------- GPU Allocation End Here -------------------
  
  
  // -------------- opacity Matrix kernel Map ----------

	int *map;
	hipMalloc(&map,sizeof(int)*3*100000000);

	int size_of_block =  ceil((1.0*V)/1024);
	map_create_kernel<<<size_of_block,1024>>>(map,GpuOpacity,V);
  //----------------------------------------------------------




	// ----------- kernel launch to create opacity Matrix ---------

  long int blockMeshes=ceil(((long int)V*100*100)/1024.0);
  opacity_kernel<<<blockMeshes,1024>>>(GpuOpacity,GpuFrameSizeX,GpuFrameSizeY,GpuGlobalCoordinatesX,GpuGlobalCoordinatesY,Opacity_matrix,V,frameSizeX,frameSizeY);
  //--------------------------------------------------------------

  // ------  kernel launch to create final matrix ----------

  long int  blockScene=ceil((1.0*frameSizeX*frameSizeY)/1024);
  scene_kernel<<<blockScene,1024>>>(map,Opacity_matrix,GpuMesh,GpuFrameSizeX,GpuFrameSizeY,GpuGlobalCoordinatesX,GpuGlobalCoordinatesY,frameSizeX,frameSizeY);
  // -----------------------------------------------------

	hipFree(GpuMesh);
	hipFree(GpuOpacity);
	hipFree(GpuFrameSizeX);
	hipFree(GpuFrameSizeY);
	hipFree(GpuGlobalCoordinatesX);
	hipFree(GpuGlobalCoordinatesY);

	

// Do not change anything below this comment.
// Code ends here.

  hipMemcpy(hFinalPng,Opacity_matrix,sizeof(int)*frameSizeX*frameSizeY,hipMemcpyDeviceToHost);


  hipFree(map);
  hipFree(Opacity_matrix);
	

	auto end  = std::chrono::high_resolution_clock::now () ;
	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken) ;
	//Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;

}
